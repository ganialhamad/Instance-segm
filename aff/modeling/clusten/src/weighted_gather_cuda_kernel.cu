#include "hip/hip_runtime.h"
/*
 * For licensing see accompanying LICENSE file.
 * Copyright (C) 2023 Apple Inc. All Rights Reserved.
 */

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/cuda/HIPContext.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/AccumulateType.h>

#define CUDA_NUM_THREADS 1024

template <typename scalar_t>
__global__ void weighted_gather_cuda_forward_kernel(
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,            // b x n x m
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> weights,            // b x n x m
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> feat,               // b x n_ x c
    torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> feat_new,                 // b x n x c
    const int length_old,           // n
    const int length,               // n_
    const int batch_size,           // b
    const int nbhd_size,            // m
    const int dim) {                // c

    const int b = blockIdx.z * blockDim.z + threadIdx.z;
    if (b < batch_size){
        const int i = blockIdx.y * blockDim.y + threadIdx.y;
        if (i < length){
            const int c = blockIdx.x * blockDim.x + threadIdx.x;
            if (c < dim){
                int64_t nbi;
                // calculate weighted feat
                scalar_t updt = scalar_t(0);
                #pragma unroll
                for (unsigned int ni=0; ni < nbhd_size; ++ni) {
                    nbi = nbhd_idx[b][i][ni];
                    updt += weights[b][i][ni] * feat[b][nbi][c];
                }
                feat_new[b][i][c] = updt;
            }
        }
    }
}


torch::Tensor weighted_gather_cuda_forward(
    const torch::Tensor &nbhd_idx,
    const torch::Tensor &weights,
    const torch::Tensor &feat) {

    int64_t batch_size = weights.size(0);
    int64_t length = weights.size(1);
    int64_t nbhd_size = weights.size(2);
    int64_t length_old = feat.size(1);
    int64_t dim = feat.size(2);

    int CHANNELTHREADS = min(int64_t(CUDA_NUM_THREADS), dim);
    int TOKENTHREADS = min(int64_t(CUDA_NUM_THREADS / CHANNELTHREADS), length);
    int BATCHTHREADS = max(1, CUDA_NUM_THREADS / (TOKENTHREADS * CHANNELTHREADS));

    auto feat_new = torch::zeros(
            {batch_size, length, dim}, weights.options());

    const auto stream = c10::cuda::getCurrentCUDAStream();
    const dim3 blocks(
            (dim + CHANNELTHREADS - 1) / CHANNELTHREADS,
            (length + TOKENTHREADS - 1) / TOKENTHREADS,
            (batch_size + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(CHANNELTHREADS, TOKENTHREADS, BATCHTHREADS);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(weights.scalar_type(), "weighted_gather_cuda_forward", ([&] {
                const auto nbhd_idx_a = nbhd_idx.packed_accessor32<int64_t,3,torch::DefaultPtrTraits>();
                const auto weights_a = weights.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
                const auto feat_a = feat.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
                auto feat_new_a = feat_new.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();

                weighted_gather_cuda_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                        nbhd_idx_a, weights_a, feat_a, feat_new_a,
                        length_old, length, batch_size, nbhd_size, dim);
                }));
    return feat_new;
}


template <typename scalar_t>
__global__ void weighted_gather_cuda_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> d_feat_new,
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> d_feat,
    const int length_old,           // n_
    const int length,               // n
    const int batch_size,           // b
    const int nbhd_size,            // m
    const int dim,                  // c
    const size_t d_feat_numel) {

    const int b = blockIdx.z * blockDim.z + threadIdx.z;
    if (b < batch_size){
        const int i = blockIdx.y * blockDim.y + threadIdx.y;
        if (i < length){
            const int c = blockIdx.x * blockDim.x + threadIdx.x;
            if (c < dim){
                int64_t nbi;
                size_t index;
                #pragma unroll
                for (unsigned int ni=0; ni < nbhd_size; ++ni) {
                    nbi = nbhd_idx[b][i][ni];
                    index = b*d_feat.stride(0) + nbi*d_feat.stride(1) + c;
                    at::native::fastAtomicAdd(d_feat.data(), index, d_feat_numel, d_feat_new[b][i][c] * weights[b][i][ni], true);
                    // atomicAdd(&(d_feat[b][nbi][c]), updt); // avoid race condition
                }
            }
        }
    }
}

template <typename scalar_t>
__global__ void weighted_gather_weights_cuda_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> d_feat_new,
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> feat,
    torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> d_weights,
    const int length_old,           // n_
    const int length,               // n
    const int batch_size,           // b
    const int nbhd_size,            // m
    const int dim) {                // c

    const int b = blockIdx.z * blockDim.z + threadIdx.z;
    if (b < batch_size){
        const int i = blockIdx.y * blockDim.y + threadIdx.y;
        if (i < length){
            const int ni = blockIdx.x * blockDim.x + threadIdx.x;
            if (ni < nbhd_size){
                int64_t nbi = nbhd_idx[b][i][ni];
                scalar_t updt = scalar_t(0);
                #pragma unroll
                for (unsigned int c=0; c < dim; ++c) {
                    // calculate d_weights = feat * d_feat_new
                    updt += feat[b][nbi][c] * d_feat_new[b][i][c];
                }
                d_weights[b][i][ni] = updt;
            }
        }
    }
}

std::vector<torch::Tensor> weighted_gather_cuda_backward(
    const torch::Tensor &d_feat_new,
    const torch::Tensor &nbhd_idx,
    const torch::Tensor &weights,
    const torch::Tensor &feat) {

    int64_t batch_size = weights.size(0);
    int64_t length = weights.size(1);
    int64_t nbhd_size = weights.size(2);
    int64_t length_old = feat.size(1);
    int64_t dim = feat.size(2);

    int CHANNELTHREADS = min(int64_t(CUDA_NUM_THREADS), dim);
    int TOKENTHREADS = min(int64_t(CUDA_NUM_THREADS / CHANNELTHREADS), length);
    int BATCHTHREADS = max(1, CUDA_NUM_THREADS / (TOKENTHREADS* CHANNELTHREADS));

    int NBHDTHREADS = min(int64_t(CUDA_NUM_THREADS), nbhd_size);
    int TOKENTHREADS_NB = min(int64_t(CUDA_NUM_THREADS / NBHDTHREADS), length);
    int BATCHTHREADS_NB = max(1, CUDA_NUM_THREADS / (TOKENTHREADS_NB* NBHDTHREADS));

    auto d_weights = torch::zeros_like(weights);
    auto d_feat = torch::zeros_like(feat);

    const auto stream = c10::cuda::getCurrentCUDAStream();

    const dim3 blocks(
            (dim + CHANNELTHREADS - 1) / CHANNELTHREADS,
            (length + TOKENTHREADS - 1) / TOKENTHREADS,
            (batch_size + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(CHANNELTHREADS, TOKENTHREADS, BATCHTHREADS);

    const dim3 blocks_nb(
            (nbhd_size + NBHDTHREADS - 1) / NBHDTHREADS,
            (length + TOKENTHREADS_NB - 1) / TOKENTHREADS_NB,
            (batch_size + BATCHTHREADS_NB - 1) / BATCHTHREADS_NB);
    const dim3 threads_nb(NBHDTHREADS, TOKENTHREADS_NB, BATCHTHREADS_NB);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(weights.scalar_type(), "weighted_gather_cuda_backward", ([&] {
                const auto d_feat_new_a = d_feat_new.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
                const auto nbhd_idx_a = nbhd_idx.packed_accessor32<int64_t,3,torch::DefaultPtrTraits>();
                const auto weights_a = weights.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
                const auto feat_a = feat.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
                auto d_weights_a = d_weights.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
                auto d_feat_a = d_feat.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();

                const size_t d_feat_numel = d_feat.numel();
                weighted_gather_cuda_backward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                        d_feat_new_a, nbhd_idx_a, weights_a, d_feat_a,
                        length_old, length, batch_size, nbhd_size, dim, d_feat_numel);
                weighted_gather_weights_cuda_backward_kernel<scalar_t><<<blocks_nb, threads_nb, 0, stream>>>(
                        d_feat_new_a, nbhd_idx_a, feat_a, d_weights_a,
                        length_old, length, batch_size, nbhd_size, dim);
                }));

    return {d_weights, d_feat};
}
